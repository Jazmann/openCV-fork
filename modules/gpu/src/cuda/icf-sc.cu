#include "hip/hip_runtime.h"
/*M///////////////////////////////////////////////////////////////////////////////////////
//
//  IMPORTANT: READ BEFORE DOWNLOADING, COPYING, INSTALLING OR USING.
//
//  By downloading, copying, installing or using the software you agree to this license.
//  If you do not agree to this license, do not download, install,
//  copy or use the software.
//
//
//                           License Agreement
//                For Open Source Computer Vision Library
//
// Copyright (C) 2000-2008, Intel Corporation, all rights reserved.
// Copyright (C) 2008-2012, Willow Garage Inc., all rights reserved.
// Third party copyrights are property of their respective owners.
//
// Redistribution and use in source and binary forms, with or without modification,
// are permitted provided that the following conditions are met:
//
//   * Redistribution's of source code must retain the above copyright notice,
//     this list of conditions and the following disclaimer.
//
//   * Redistribution's in binary form must reproduce the above copyright notice,
//     this list of conditions and the following disclaimer in the documentation
//     and/or other materials provided with the distribution.
//
//   * The name of the copyright holders may not be used to endorse or promote products
//     derived from this software without specific prior written permission.
//
// This software is provided by the copyright holders and contributors "as is" and
// any express or implied warranties, including, but not limited to, the implied
// warranties of merchantability and fitness for a particular purpose are disclaimed.
// In no event shall the Intel Corporation or contributors be liable for any direct,
// indirect, incidental, special, exemplary, or consequential damages
// (including, but not limited to, procurement of substitute goods or services;
// loss of use, data, or profits; or business interruption) however caused
// and on any theory of liability, whether in contract, strict liability,
// or tort (including negligence or otherwise) arising in any way out of
// the use of this software, even if advised of the possibility of such damage.
//
//M*/

#include <opencv2/gpu/device/common.hpp>

#include <icf.hpp>
#include <float.h>
#include <stdio.h>

namespace cv { namespace gpu { namespace device {
namespace icf {

    // ToDo: use textures or uncached load instruction.
    __global__ void magToHist(const uchar* __restrict__ mag,
                              const float* __restrict__ angle, const int angPitch,
                                    uchar* __restrict__ hog,   const int hogPitch, const int fh)
    {
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;

        const int bin = (int)(angle[y * angPitch + x]);
        const uchar val = mag[y * hogPitch + x];
        hog[((fh * bin) + y) * hogPitch + x] = val;
    }

    void fillBins(cv::gpu::PtrStepSzb hogluv, const cv::gpu::PtrStepSzf& nangle,
                  const int fw,  const int fh, const int bins, hipStream_t stream )
    {
        const uchar* mag = (const uchar*)hogluv.ptr(fh * bins);
        uchar* hog = (uchar*)hogluv.ptr();
        const float* angle = (const float*)nangle.ptr();

        dim3 block(32, 8);
        dim3 grid(fw / 32, fh / 8);

        magToHist<<<grid, block, 0, stream>>>(mag, angle, nangle.step / sizeof(float), hog, hogluv.step, fh);
        if (!stream)
        {
            cudaSafeCall( hipGetLastError() );
            cudaSafeCall( hipDeviceSynchronize() );
        }
    }

    __device__ __forceinline__ float overlapArea(const Detection &a, const Detection &b)
    {
        int w = ::min(a.x + a.w, b.x + b.w) - ::max(a.x, b.x);
        int h = ::min(a.y + a.h, b.y + b.h) - ::max(a.y, b.y);

        return (w < 0 || h < 0)? 0.f : (float)(w * h);
    }

    texture<uint4,  hipTextureType2D, hipReadModeElementType> tdetections;

    __global__ void overlap(const uint* n, uchar* overlaps)
    {
        const int idx = threadIdx.x;
        const int total = *n;

        for (int i = idx + 1; i < total; i += 192)
        {
            const uint4 _a = tex2D(tdetections, i, 0);
            const Detection& a = *((Detection*)(&_a));
            bool excluded = false;

            for (int j = i + 1; j < total; ++j)
            {
                const uint4 _b = tex2D(tdetections, j, 0);
                const Detection& b = *((Detection*)(&_b));
                float ovl = overlapArea(a, b) / ::min(a.w * a.h, b.w * b.h);

                if (ovl > 0.65f)
                {
                    int suppessed = (a.confidence > b.confidence)? j : i;
                    overlaps[suppessed] = 1;
                    excluded = excluded || (suppessed == i);
                }

            #if __CUDA_ARCH__ >= 120
                if (__all(excluded)) break;
            #endif
            }
        }
    }

    __global__ void collect(const uint* n, uchar* overlaps, uint* ctr, uint4* suppressed)
    {
        const int idx = threadIdx.x;
        const int total = *n;

        for (int i = idx; i < total; i += 192)
        {
            if (!overlaps[i])
            {
                int oidx = atomicInc(ctr, 50);
                suppressed[oidx] = tex2D(tdetections, i + 1, 0);
            }
        }
    }

    void suppress(const PtrStepSzb& objects, PtrStepSzb overlaps, PtrStepSzi ndetections,
        PtrStepSzb suppressed, hipStream_t stream)
    {
        int block = 192;
        int grid = 1;

        hipChannelFormatDesc desc = hipCreateChannelDesc<uint4>();
        size_t offset;
        cudaSafeCall( hipBindTexture2D(&offset, tdetections, objects.data, desc, objects.cols / sizeof(uint4), objects.rows, objects.step));

        overlap<<<grid, block>>>((uint*)ndetections.ptr(0), (uchar*)overlaps.ptr(0));
        collect<<<grid, block>>>((uint*)ndetections.ptr(0), (uchar*)overlaps.ptr(0), (uint*)suppressed.ptr(0), ((uint4*)suppressed.ptr(0)) + 1);

        if (!stream)
        {
            cudaSafeCall( hipGetLastError());
            cudaSafeCall( hipDeviceSynchronize());
        }
    }

    template<typename Policy>
    struct PrefixSum
    {
    __device static void apply(float& impact)
        {
    #if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
    #pragma unroll
            // scan on shuffl functions
            for (int i = 1; i < Policy::WARP; i *= 2)
            {
                const float n = __shfl_up(impact, i, Policy::WARP);

                if (threadIdx.x >= i)
                    impact += n;
            }
    #else
            __shared__ volatile float ptr[Policy::STA_X * Policy::STA_Y];

            const int idx = threadIdx.y * Policy::STA_X + threadIdx.x;

            ptr[idx] = impact;

            if ( threadIdx.x >=  1) ptr [idx ] = (ptr [idx -  1] + ptr [idx]);
            if ( threadIdx.x >=  2) ptr [idx ] = (ptr [idx -  2] + ptr [idx]);
            if ( threadIdx.x >=  4) ptr [idx ] = (ptr [idx -  4] + ptr [idx]);
            if ( threadIdx.x >=  8) ptr [idx ] = (ptr [idx -  8] + ptr [idx]);
            if ( threadIdx.x >= 16) ptr [idx ] = (ptr [idx - 16] + ptr [idx]);

            impact = ptr[idx];
    #endif
        }
    };

    texture<int,  hipTextureType2D, hipReadModeElementType> thogluv;

    template<bool isUp>
    __device__ __forceinline__ float rescale(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = (scaledRect.z - scaledRect.x) * (scaledRect.w - scaledRect.y);

        const float expected_new_area = farea * relScale * relScale;
        float approx = (sarea == 0)? 1: __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<>
    __device__ __forceinline__ float rescale<true>(const Level& level, Node& node)
    {
        uchar4& scaledRect = node.rect;
        float relScale = level.relScale;
        float farea = scaledRect.z * scaledRect.w;

        // rescale
        scaledRect.x = __float2int_rn(relScale * scaledRect.x);
        scaledRect.y = __float2int_rn(relScale * scaledRect.y);
        scaledRect.z = __float2int_rn(relScale * scaledRect.z);
        scaledRect.w = __float2int_rn(relScale * scaledRect.w);

        float sarea = scaledRect.z * scaledRect.w;

        const float expected_new_area = farea * relScale * relScale;
        float approx = __fdividef(sarea, expected_new_area);

        float rootThreshold = (node.threshold & 0x0FFFFFFFU) * approx * level.scaling[(node.threshold >> 28) > 6];

        return rootThreshold;
    }

    template<bool isUp>
    __device__ __forceinline__ int get(int x, int y, uchar4 area)
    {
        int a = tex2D(thogluv, x + area.x, y + area.y);
        int b = tex2D(thogluv, x + area.z, y + area.y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x + area.x, y + area.w);

        return (a - b + c - d);
    }

    template<>
    __device__ __forceinline__ int get<true>(int x, int y, uchar4 area)
    {
        x += area.x;
        y += area.y;
        int a = tex2D(thogluv, x, y);
        int b = tex2D(thogluv, x + area.z, y);
        int c = tex2D(thogluv, x + area.z, y + area.w);
        int d = tex2D(thogluv, x, y + area.w);

        return (a - b + c - d);
    }

    texture<float2,  hipTextureType2D, hipReadModeElementType> troi;

template<typename Policy>
template<bool isUp>
__device void CascadeInvoker<Policy>::detect(Detection* objects, const uint ndetections, uint* ctr, const int downscales) const
{
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int x = blockIdx.x;

    // load Lavel
    __shared__ Level level;

    // check POI
    __shared__ volatile char roiCache[Policy::STA_Y];

    if (!threadIdx.y && !threadIdx.x)
        ((float2*)roiCache)[threadIdx.x] = tex2D(troi, blockIdx.y, x);

    __syncthreads();

    if (!roiCache[threadIdx.y]) return;

    if (!threadIdx.x)
        level = levels[downscales + blockIdx.z];

    if(x >= level.workRect.x || y >= level.workRect.y) return;

    int st = level.octave * level.step;
    const int stEnd = st + level.step;

    const int hogluvStep = gridDim.y * Policy::STA_Y;
    float confidence = 0.f;
    for(; st < stEnd; st += Policy::WARP)
    {
        const int nId = (st + threadIdx.x) * 3;

        Node node = nodes[nId];

        float threshold = rescale<isUp>(level, node);
        int sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        int next = 1 + (int)(sum >= threshold);

        node = nodes[nId + next];
        threshold = rescale<isUp>(level, node);
        sum = get<isUp>(x, y + (node.threshold >> 28) * hogluvStep, node.rect);

        const int lShift = (next - 1) * 2 + (int)(sum >= threshold);
        float impact = leaves[(st + threadIdx.x) * 4 + lShift];

        PrefixSum<Policy>::apply(impact);
        confidence += impact;

    #if __CUDA_ARCH__ >= 120
        if(__any((confidence <= stages[(st + threadIdx.x)]))) st += 2048;
    #endif
    }

    if(!threadIdx.x && st == stEnd &&  ((confidence - FLT_EPSILON) >= 0))
    {
        int idx = atomicInc(ctr, ndetections);
        objects[idx] = Detection(__float2int_rn(x * Policy::SHRINKAGE),
            __float2int_rn(y * Policy::SHRINKAGE), level.objSize.x, level.objSize.y, confidence);
    }
}

template<typename Policy, bool isUp>
__global__ void soft_cascade(const CascadeInvoker<Policy> invoker, Detection* objects, const uint n, uint* ctr, const int downs)
{
    invoker.template detect<isUp>(objects, n, ctr, downs);
}

template<typename Policy>
void CascadeInvoker<Policy>::operator()(const PtrStepSzb& roi, const PtrStepSzi& hogluv,
    PtrStepSz<uchar4> objects, const int downscales, const hipStream_t& stream) const
{
    int fw = roi.rows;
    int fh = roi.cols;

    dim3 grid(fw, fh / Policy::STA_Y, downscales);

    uint* ctr = (uint*)(objects.ptr(0));
    Detection* det = ((Detection*)objects.ptr(0)) + 1;
    uint max_det = objects.cols / sizeof(Detection);

    hipChannelFormatDesc desc = hipCreateChannelDesc<int>();
    cudaSafeCall( hipBindTexture2D(0, thogluv, hogluv.data, desc, hogluv.cols, hogluv.rows, hogluv.step));

    hipChannelFormatDesc desc_roi = hipCreateChannelDesc<typename Policy::roi_type>();
    cudaSafeCall( hipBindTexture2D(0, troi, roi.data, desc_roi, roi.cols / Policy::STA_Y, roi.rows, roi.step));

    const CascadeInvoker<Policy> inv = *this;

    soft_cascade<Policy, false><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, 0);
    cudaSafeCall( hipGetLastError());

    grid = dim3(fw, fh / Policy::STA_Y, scales - downscales);
    soft_cascade<Policy, true><<<grid, Policy::block(), 0, stream>>>(inv, det, max_det, ctr, downscales);

    if (!stream)
    {
        cudaSafeCall( hipGetLastError());
        cudaSafeCall( hipDeviceSynchronize());
    }
}

template void CascadeInvoker<GK107PolicyX4>::operator()(const PtrStepSzb& roi, const PtrStepSzi& hogluv,
    PtrStepSz<uchar4> objects, const int downscales, const hipStream_t& stream) const;

}
}}}
